#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <iostream>
#include "gputimer.h"

#include "Constants.h"

#define funcCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

__global__ void matrixMult(int* A, int* B, int* C, int nT)
{
	int CValue = 0;

	int Row = blockIdx.y * TILE_DIM + threadIdx.y;
	int Col = blockIdx.x * TILE_DIM + threadIdx.x;

	__shared__ int As[TILE_DIM][TILE_DIM];
	__shared__ int Bs[TILE_DIM][TILE_DIM];

	for (int k = 0; k < nT; k++) {
		if (k * TILE_DIM + threadIdx.x < MATRIX_DIM && Row < MATRIX_DIM)
			As[threadIdx.y][threadIdx.x] = A[Row * MATRIX_DIM + k * TILE_DIM + threadIdx.x];
		else
			As[threadIdx.y][threadIdx.x] = 0.0;

		if (k*TILE_DIM + threadIdx.y < MATRIX_DIM && Col < MATRIX_DIM)
			Bs[threadIdx.y][threadIdx.x] = B[(k * TILE_DIM + threadIdx.y) * MATRIX_DIM + Col];
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0;

		__syncthreads();

		for (int n = 0; n < TILE_DIM; n++) {
			CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
		}

		__syncthreads();
	}

	C[threadIdx.x + TILE_DIM] = CValue;
}

// Inicializa una matriz con valores aleatorios
void randomInit(int *data, int size)
{
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; ++i)
	{
		data[i] = rand() % 20 - 10;  // random between -10 and 10
	}
}

// Inicializa una matriz con ceros
void zeroInit(int *data, int size)
{
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; ++i)
	{
		data[i] = 0;
	}
}

int main(int argc, char ** argv) {
	GpuTimer timer;
	int * h_A; // The A matrix
	int * h_B; // The B matrix
	int * h_C; // The output C matrix
	int * d_A;
	int * d_B;
	int * d_C;
	int numTiles = (int)ceil((float)TILE_DIM / TILE_DIM);

	printf("Matrix dimension: %d x %d elements\n", MATRIX_DIM, MATRIX_DIM);
	printf("Block dimension: %d x %d threads\n", TILE_DIM, TILE_DIM);
	printf("Grid dimension: %d x %d blocks\n", numTiles, numTiles);
	printf("Grid dimension: %d x %d threads\n", numTiles*TILE_DIM, numTiles*TILE_DIM);
	printf("Number of iterations: %d\n", NITERS*NITERS2);

	int memoryPos = TILE_DIM * TILE_DIM;
	int memorySize = sizeof(int) * memoryPos;

	h_A = (int *)malloc(memorySize);
	h_B = (int *)malloc(memorySize);
	randomInit(h_A, memoryPos);
	randomInit(h_B, memoryPos);

	h_C = (int *)malloc(memorySize);
	zeroInit(h_C, memoryPos);

	printf("Matrices were initialized\n");

	// Allocating GPU memory
	funcCheck(hipMalloc((void **)&d_A, memorySize));
	funcCheck(hipMalloc((void **)&d_B, memorySize));
	funcCheck(hipMalloc((void **)&d_C, memorySize));

	// Copy memory to the GPU 
	funcCheck(hipMemcpy(d_A, h_A, memorySize, hipMemcpyHostToDevice));
	funcCheck(hipMemcpy(d_B, h_B, memorySize, hipMemcpyHostToDevice));

	// Initialize the grid and block dimensions 
	dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
	dim3 dimGrid(numTiles, numTiles, 1);

	//@@ Launch the GPU Kernel here
	timer.Start();
	for (int niter = 0; niter < NITERS*NITERS2; niter++) {
		matrixMult << <dimGrid, dimBlock >> > (d_A, d_B, d_C, numTiles);
	}
	timer.Stop();
	printf("Tiled & coalescend kernel processing time: %f millisec.\n", timer.Elapsed());

	// Copy the results in GPU memory back to the CPU    
	funcCheck(hipMemcpy(h_C, d_C, memorySize, hipMemcpyDeviceToHost));

	// Free the GPU memory
	funcCheck(hipFree(d_A));
	funcCheck(hipFree(d_B));
	funcCheck(hipFree(d_C));

	free(h_A);
	free(h_B);
	free(h_C);

	getchar();

	return 0;
}