#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "PPMImage.h"
#include "CudaException.h"

#define COLOR_FILTER_R 1.0
#define COLOR_FILTER_G 0.0
#define COLOR_FILTER_B 0.75

// Kernel que procesa cada color del pixel
__global__ void colorFilterKernel(unsigned char *data, long dataSize)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int col = i % 3;
	data[i] = data[i] * (col == 0 ? COLOR_FILTER_R : (col == 1 ? COLOR_FILTER_G : COLOR_FILTER_B));
}

// Funci�n que procesa la imagen para hacer un filtro de color
hipError_t colorFilterWithCuda(PPMImage *image)
{
	hipError_t cudaStatus;
	unsigned char *imageData_d;

	try {
		// Elegimos la primera GPU (No tenemos en cuenta sistemas multi GPU)
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) throw new CudaException("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", cudaStatus);

		cudaStatus = hipMalloc((void**)&imageData_d, image->dataSize);
		if (cudaStatus != hipSuccess) throw new CudaException("hipMalloc failed!", cudaStatus);

		cudaStatus = hipMemcpy(imageData_d, &image->data, image->dataSize, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) throw new CudaException("hipMemcpy failed!", cudaStatus);

		int numBlocks_h = image->dataSize / 32;
		colorFilterKernel << <numBlocks_h, 32 >> > (imageData_d, image->dataSize);

		// Detectamos el error de ejecuci�n
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) throw new CudaException("colorFilterKernel launch failed!", cudaStatus);

		// Espera a que termine todas las ejecuciones y detecta errores
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) throw new CudaException("hipDeviceSynchronize returned error code %d after launching addKernel!", cudaStatus);

		// Copiamos el vector resultado a RAM
		cudaStatus = hipMemcpy(image->data, imageData_d, image->dataSize, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) throw new CudaException("hipMemcpy failed!", cudaStatus);
	}
	catch (CudaException e) {
		hipFree(imageData_d);
		throw e;
	}

	return cudaStatus;
}

int main()
{
	try {
		std::cout << "Leyendo imagen... ";

		PPMImage sourceImage;
		loadPpmImage("lena.ppm", &sourceImage);

		std::cout << "correcto!" << std::endl;
		std::cout << "Procesando imagen... ";

		hipError_t cudaStatus = colorFilterWithCuda(&sourceImage);
		if (cudaStatus != hipSuccess) throw new CudaException("Ha habido un error al convertir la imagen con CUDA.", cudaStatus);

		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) throw new CudaException("Error al reiniciar el estado del dispositivo.", cudaStatus);

		std::cout << "correcto!" << std::endl;
		std::cout << "Guardando imagen... ";

		savePpmImage("lenaFiltered.ppm", &sourceImage);

		std::cout << "correcto!" << std::endl;

		return 0;
	}
	catch (std::exception e) {
		std::cout << "error!" << std::endl;
		std::cerr << e.what() << std::endl;
		std::cout << "Pulsa ENTER para salir..." << std::endl;
		int x = std::cin.get();

		return 1;
	}
}
